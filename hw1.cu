/* compile with: nvcc -O3 hw1.cu -o hw1 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

///////////////////////////////////////////////// DO NOT CHANGE ///////////////////////////////////////
#define IMG_HEIGHT 256
#define IMG_WIDTH 256
//#define N_IMAGES 10000
#define N_IMAGES 500

#define NUM_THREADS 256


typedef unsigned char uchar;

#define CUDA_CHECK(f) do {                                                                  \
    hipError_t e = f;                                                                      \
    if (e != hipSuccess) {                                                                 \
        printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));    \
        exit(1);                                                                            \
    }                                                                                       \
} while (0)

#define SQR(a) ((a) * (a))

void process_image(uchar *img_in, uchar *img_out) {
    int histogram[256] = { 0 };
    for (int i = 0; i < IMG_WIDTH * IMG_HEIGHT; i++) {
        histogram[img_in[i]]++;
    }

    int cdf[256] = { 0 };
    int hist_sum = 0;
    for (int i = 0; i < 256; i++) {
        hist_sum += histogram[i];
        cdf[i] = hist_sum;
    }

    int cdf_min = 0;
    for (int i = 0; i < 256; i++) {
        if (cdf[i] != 0) {
            cdf_min = cdf[i];
            break;
        }
    }

    uchar map[256] = { 0 };
    for (int i = 0; i < 256; i++) {
        int map_value = (float)(cdf[i] - cdf_min) / (IMG_WIDTH * IMG_HEIGHT - cdf_min) * 255;
        map[i] = (uchar)map_value;
    }

    for (int i = 0; i < IMG_WIDTH * IMG_HEIGHT; i++) {
        img_out[i] = map[img_in[i]];
    }
}

double static inline get_time_msec(void) {
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec * 1e+3 + t.tv_usec * 1e-3;
}

long long int distance_sqr_between_image_arrays(uchar *img_arr1, uchar *img_arr2) {
    long long int distance_sqr = 0;
    for (int i = 0; i < N_IMAGES * IMG_WIDTH * IMG_HEIGHT; i++) {
        distance_sqr += SQR(img_arr1[i] - img_arr2[i]);
    }
    return distance_sqr;
}
///////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ int arr_min(int arr[], int arr_size) {//return the min
	int tid = threadIdx.x;
	int half_length = arr_size / 2;

	while (half_length >= 1) {
		for (int i = tid; i < half_length; i += blockDim.x) {
			if(arr[tid + i] < arr[i]) arr[i] = arr[tid + i];
		}
		__syncthreads();
		half_length /= 2;
	}
    return arr[0]; //TODO
}

// this function implements the Kiggle-Stone algorithm
__device__ void prefix_sum(int arr[], int arr_size, int histogram[]) {

	int tbsize = blockDim.x;
	int tid = threadIdx.x;
	int inc;
 
 	for (int stride = 1; stride < tbsize; stride *= 2) {
	
		if (tid >= arr_size)
			continue;

 		if (tid >= stride) {
 		inc = arr[tid - stride];
 		}
 		__syncthreads();

 		if (tid >= stride) {
			arr[tid] += inc;
 		}

 		__syncthreads();
 	}

    return;
}

__device__ void mapCalc(int map[], int min, int cdf[]) {
	int id = threadIdx.x;
	int map_value = ((double)(cdf[id] - min)/(IMG_WIDTH * IMG_HEIGHT - min)) * 255;
    map[id] = map_value;
}

__global__ void process_image_kernel(uchar *in, uchar *out) {

    __shared__ int l_histogram[256];
    __shared__ int l_cdf[256];
    __shared__ int map[256];
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int tbsize = blockDim.x;

	// zero histogram
	l_histogram[tid] = 0;

	__syncthreads();

	for(int i = tid; i < IMG_WIDTH * IMG_HEIGHT; i += tbsize)
		atomicAdd(&l_histogram[in[(IMG_WIDTH * IMG_HEIGHT)*bid + i]], 1);

	__syncthreads();

	// prepare the cdf array in advance
	l_cdf[tid] = l_histogram[tid];

	__syncthreads();

	prefix_sum(l_cdf, 256, l_histogram);

	__syncthreads();

    int min = arr_min(l_histogram, 256);

	__syncthreads();

	mapCalc(map, min, l_cdf);

	__syncthreads();
   

    for(int i = tid; i < IMG_WIDTH * IMG_HEIGHT; i += tbsize) {
		out[(IMG_WIDTH * IMG_HEIGHT)*bid + i] =
			map[in[(IMG_WIDTH * IMG_HEIGHT)*bid + i]];
    }

	__syncthreads();

    return ; //TODO
}

int main() {
///////////////////////////////////////////////// DO NOT CHANGE ///////////////////////////////////////
    uchar *images_in;
    uchar *images_out_cpu; //output of CPU computation. In CPU memory.
    uchar *images_out_gpu_serial; //output of GPU task serial computation. In CPU memory.
    uchar *images_out_gpu_bulk; //output of GPU bulk computation. In CPU memory.
    CUDA_CHECK( hipHostAlloc(&images_in, N_IMAGES * IMG_HEIGHT * IMG_WIDTH, 0) );
    CUDA_CHECK( hipHostAlloc(&images_out_cpu, N_IMAGES * IMG_HEIGHT * IMG_WIDTH, 0) );
    CUDA_CHECK( hipHostAlloc(&images_out_gpu_serial, N_IMAGES * IMG_HEIGHT * IMG_WIDTH, 0) );
    CUDA_CHECK( hipHostAlloc(&images_out_gpu_bulk, N_IMAGES * IMG_HEIGHT * IMG_WIDTH, 0) );

    /* instead of loading real images, we'll load the arrays with random data */
    srand(0);
    for (long long int i = 0; i < N_IMAGES * IMG_WIDTH * IMG_HEIGHT; i++) {
        images_in[i] = rand() % 256;
    }

    double t_start, t_finish;

    // CPU computation. For reference. Do not change
    printf("\n=== CPU ===\n");
    t_start = get_time_msec();
    for (int i = 0; i < N_IMAGES; i++) {
        uchar *img_in = &images_in[i * IMG_WIDTH * IMG_HEIGHT];
        uchar *img_out = &images_out_cpu[i * IMG_WIDTH * IMG_HEIGHT];
		process_image(img_in, img_out);
    }
    t_finish = get_time_msec();
    printf("total time %f [msec]\n", t_finish - t_start);

    long long int distance_sqr;
///////////////////////////////////////////////////////////////////////////////////////////////////////////
	uchar *image_in;
	uchar *image_out;

    // GPU task serial computation
    printf("\n=== GPU Task Serial ===\n"); //Do not change

    //TODO: allocate GPU memory for a single input image and a single output image
    CUDA_CHECK( hipMalloc((void **)&image_in, IMG_HEIGHT * IMG_WIDTH) );
    CUDA_CHECK( hipMalloc((void **)&image_out, IMG_HEIGHT * IMG_WIDTH) );

    t_start = get_time_msec(); //Do not change

    //TODO: in a for loop:
    for (int i=0; i < N_IMAGES; i++) {
		// Copying src image from the input images
		CUDA_CHECK(hipMemcpy(image_in, &images_in[i * IMG_WIDTH*IMG_HEIGHT], IMG_WIDTH*IMG_HEIGHT, hipMemcpyDefault));

		process_image_kernel <<< 1, NUM_THREADS >>> (image_in, image_out);  

		CUDA_CHECK(hipDeviceSynchronize());

		CUDA_CHECK(hipMemcpy(&images_out_gpu_serial[i * IMG_HEIGHT*IMG_WIDTH], image_out, IMG_WIDTH*IMG_HEIGHT, hipMemcpyDefault));
    }

	hipFree(image_in);
	hipFree(image_out);

    t_finish = get_time_msec(); //Do not change
    distance_sqr = distance_sqr_between_image_arrays(images_out_cpu, images_out_gpu_serial); // Do not change
    printf("total time %f [msec]  distance from baseline %lld (should be zero)\n", t_finish - t_start, distance_sqr); //Do not change

    // GPU bulk
    printf("\n=== GPU Bulk ===\n"); //Do not change
    //TODO: allocate GPU memory for a all input images and all output images
	CUDA_CHECK( hipMalloc((void **)&image_in, N_IMAGES * IMG_HEIGHT * IMG_WIDTH) );
    CUDA_CHECK( hipMalloc((void **)&image_out, N_IMAGES * IMG_HEIGHT * IMG_WIDTH) );

    //TODO: copy all input images from images_in to the GPU memory you allocated
    t_start = get_time_msec(); //Do not change
	CUDA_CHECK(hipMemcpy(image_in, images_in, N_IMAGES*IMG_WIDTH*IMG_HEIGHT, hipMemcpyDefault));

    //TODO: invoke a kernel with N_IMAGES threadblocks, each working on a different image
	process_image_kernel <<< N_IMAGES, NUM_THREADS >>> (image_in, image_out);
	CUDA_CHECK(hipDeviceSynchronize());

    //TODO: copy output images from GPU memory to images_out_gpu_bulk
	CUDA_CHECK(hipMemcpy(images_out_gpu_bulk, image_out, N_IMAGES*IMG_WIDTH*IMG_HEIGHT, hipMemcpyDefault));

    t_finish = get_time_msec(); //Do not change

	hipFree(image_in);
	hipFree(image_out);

    distance_sqr = distance_sqr_between_image_arrays(images_out_cpu, images_out_gpu_bulk); // Do not change
    printf("total time %f [msec]  distance from baseline %lld (should be zero)\n", t_finish - t_start, distance_sqr); //Do not chhange

    return 0;
}
