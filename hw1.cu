/* compile with: nvcc -O3 hw1.cu -o hw1 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

///////////////////////////////////////////////// DO NOT CHANGE ///////////////////////////////////////
#define IMG_HEIGHT 256
#define IMG_WIDTH 256
//#define N_IMAGES 10000
#define N_IMAGES 1

#define NUM_THREADS 256


typedef unsigned char uchar;

#define CUDA_CHECK(f) do {                                                                  \
    hipError_t e = f;                                                                      \
    if (e != hipSuccess) {                                                                 \
        printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));    \
        exit(1);                                                                            \
    }                                                                                       \
} while (0)

#define SQR(a) ((a) * (a))

void process_image(uchar *img_in, uchar *img_out) {
    int histogram[256] = { 0 };
    for (int i = 0; i < IMG_WIDTH * IMG_HEIGHT; i++) {
        histogram[img_in[i]]++;
    }

    int cdf[256] = { 0 };
    int hist_sum = 0;
    for (int i = 0; i < 256; i++) {
        hist_sum += histogram[i];
        cdf[i] = hist_sum;
    }

    int cdf_min = 0;
    for (int i = 0; i < 256; i++) {
        if (cdf[i] != 0) {
            cdf_min = cdf[i];
            break;
        }
    }

    uchar map[256] = { 0 };
    for (int i = 0; i < 256; i++) {
        int map_value = (float)(cdf[i] - cdf_min) / (IMG_WIDTH * IMG_HEIGHT - cdf_min) * 255;
        map[i] = (uchar)map_value;
    }

    for (int i = 0; i < IMG_WIDTH * IMG_HEIGHT; i++) {
        img_out[i] = map[img_in[i]];
    }
}

double static inline get_time_msec(void) {
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec * 1e+3 + t.tv_usec * 1e-3;
}

long long int distance_sqr_between_image_arrays(uchar *img_arr1, uchar *img_arr2) {
    long long int distance_sqr = 0;
    for (int i = 0; i < N_IMAGES * IMG_WIDTH * IMG_HEIGHT; i++) {
        distance_sqr += SQR(img_arr1[i] - img_arr2[i]);
    }
    return distance_sqr;
}
///////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ int arr_min(int arr[], int arr_size) {//return the min
	int tid = threadIdx.x;
	int half_length = arr_size / 2;

	while (half_length >= 1) {
		for (int i = tid; i < half_length; i += blockDim.x) {
			if(arr[tid + i] < arr[i]) arr[i] = arr[tid + i];
		}
		__syncthreads();
		half_length /= 2;
	}
    return arr[0]; //TODO
}

// this function implements the Kiggle-Stone algorithm
__device__ void prefix_sum(int arr[], int arr_size, int histogram[]) {
 
	int tbsize = blockDim.x;
	int tid = threadIdx.x;
	int inc;
 	
 	for (int stride = 1; stride < tbsize; stride *= 2) {
		
		if (tid >= arr_size)
			continue;

 		if (tid >= stride) {
 		inc = arr[tid - stride];
 		}
 		__syncthreads();

 		if (tid >= stride) {
			arr[tid] += inc;
 		}

 		__syncthreads();
 	}

    return;
}

__device__ void mapCalc(int map[], int min, int cdf[]) {
	int id = threadIdx.x;
	int map_value = ((double)(cdf[id] - min)/(IMG_WIDTH * IMG_HEIGHT - min)) * 255;
    map[id] = map_value;
}

__global__ void process_image_kernel(uchar *in, uchar *out, int temp_histogram[], int temp_cdf[]) {

    __shared__ int l_histogram[256];
    __shared__ int l_cdf[256];
    __shared__ int map[256];
	int tid = threadIdx.x;
	int tbsize = blockDim.x;

	// zero histogram
	l_histogram[tid] = 0;

	for(int i = tid; i < IMG_WIDTH * IMG_HEIGHT; i += tbsize)
		atomicAdd(&l_histogram[in[i]], 1);

	__syncthreads();

	// for debug purposes TODO: delete
	temp_histogram[tid] = l_histogram[tid];

	// prepare the cdf array in advance
	l_cdf[tid] = l_histogram[tid];

	__syncthreads();

	prefix_sum(l_cdf, 256, l_histogram);

	// for debug purposes TODO: delete
	temp_cdf[tid] = l_cdf[tid];

	__syncthreads();

    int min = arr_min(l_histogram, 256);
    
    if(tid == 0)
    	printf("min is: %d\n", min);
	
	// for debug purposes TODO: delete
	temp_cdf[tid] = l_histogram[tid];
	
	__syncthreads();

	mapCalc(map, min, l_cdf);
	
	__syncthreads();
    

    for(int i = tid; i < IMG_WIDTH * IMG_HEIGHT; i += tbsize) {
		out[i] = map[in[i]];
    }
	
	__syncthreads();

    return ; //TODO
}

int main() {
///////////////////////////////////////////////// DO NOT CHANGE ///////////////////////////////////////
    uchar *images_in;
    uchar *images_out_cpu; //output of CPU computation. In CPU memory.
    uchar *images_out_gpu_serial; //output of GPU task serial computation. In CPU memory.
    uchar *images_out_gpu_bulk; //output of GPU bulk computation. In CPU memory.
    CUDA_CHECK( hipHostAlloc(&images_in, N_IMAGES * IMG_HEIGHT * IMG_WIDTH, 0) );
    CUDA_CHECK( hipHostAlloc(&images_out_cpu, N_IMAGES * IMG_HEIGHT * IMG_WIDTH, 0) );
    CUDA_CHECK( hipHostAlloc(&images_out_gpu_serial, N_IMAGES * IMG_HEIGHT * IMG_WIDTH, 0) );
    CUDA_CHECK( hipHostAlloc(&images_out_gpu_bulk, N_IMAGES * IMG_HEIGHT * IMG_WIDTH, 0) );

    /* instead of loading real images, we'll load the arrays with random data */
    srand(0);
    for (long long int i = 0; i < N_IMAGES * IMG_WIDTH * IMG_HEIGHT; i++) {
        images_in[i] = rand() % 256;
    }

    double t_start, t_finish;

    // CPU computation. For reference. Do not change
    printf("\n=== CPU ===\n");
    t_start = get_time_msec();
    for (int i = 0; i < N_IMAGES; i++) {
        uchar *img_in = &images_in[i * IMG_WIDTH * IMG_HEIGHT];
        uchar *img_out = &images_out_cpu[i * IMG_WIDTH * IMG_HEIGHT];
		process_image(img_in, img_out);
    }
    t_finish = get_time_msec();
    printf("total time %f [msec]\n", t_finish - t_start);

    long long int distance_sqr;
///////////////////////////////////////////////////////////////////////////////////////////////////////////
	uchar *image_in;
	uchar *image_out;

	int *temp_histogram;
	int *temp_cdf;
	int cpu_histogram[256] = { 0 };
	int cpu_cdf[256] = { 0 };
	int total_sum = 0;

    // TODO: debug, remove later
    CUDA_CHECK( hipMalloc((void **)&temp_histogram, 256 * sizeof(*temp_histogram)) );
    CUDA_CHECK( hipMalloc((void **)&temp_cdf, 256 * sizeof(*temp_cdf)) );

    // GPU task serial computation
    printf("\n=== GPU Task Serial ===\n"); //Do not change

    //TODO: allocate GPU memory for a single input image and a single output image
    CUDA_CHECK( hipMalloc((void **)&image_in, IMG_HEIGHT * IMG_WIDTH) );
    CUDA_CHECK( hipMalloc((void **)&image_out, IMG_HEIGHT * IMG_WIDTH) );

    t_start = get_time_msec(); //Do not change

    //TODO: in a for loop:
    for (int i=0; i < N_IMAGES; i++) {
		// Copying src image from the input images
		hipMemcpy(image_in, &images_in[i * IMG_WIDTH*IMG_HEIGHT], IMG_WIDTH*IMG_HEIGHT, hipMemcpyDefault);
		
		// TODO: debug, remove later
		hipMemset(temp_histogram, 0, 256 * sizeof(*temp_histogram));
		hipMemset(temp_cdf, 0, 256 * sizeof(*temp_cdf));

		process_image_kernel <<< 1, NUM_THREADS >>> (image_in, image_out, temp_histogram, temp_cdf);   

		hipDeviceSynchronize();

		// TODO: debug, remove later
		hipMemcpy(cpu_histogram, temp_histogram, 256 * sizeof(*temp_histogram), hipMemcpyDefault);
		hipMemcpy(cpu_cdf, temp_cdf, 256 * sizeof(*temp_cdf), hipMemcpyDefault);

		// Debug prints TODO: remove later
		printf ("\n\nHistogram array is as followed:\n");
		for (int m=0; m< 4; m++) {
			for (int j = 0; j < 64; j++) {
				printf("h[%d] = %d  ",m*64 + j , cpu_histogram[m*64 + j]);
				total_sum += cpu_histogram[m*64 + j];
			}

			printf("\n\n");
		}

		// TODO: delete
		printf("Total sum is: %d\n", total_sum);

		// TODO: delete
		printf ("\n\nCDF array is as followed:\n");
		for (int m=0; m < 4; m++) {
			for (int j = 0; j < 64; j++) {
				printf("h[%d] = %d  ",m*64 + j , cpu_cdf[m*64 + j]);
			}

			printf("\n\n");
		}

		hipMemcpy(&images_out_gpu_serial[i * IMG_HEIGHT*IMG_WIDTH], image_out, IMG_WIDTH*IMG_HEIGHT, hipMemcpyDefault);

		bool bad = 0;
		for(int m = 0; m < IMG_WIDTH * IMG_HEIGHT; m++) {
			if (images_out_gpu_serial[(i*IMG_WIDTH*IMG_HEIGHT) + m] != images_out_cpu[(i*IMG_WIDTH*IMG_HEIGHT) + m]) {
				printf("CPU value[%d]: %d\n",m, images_out_cpu[(i*IMG_WIDTH*IMG_HEIGHT) + m]);
				printf("GPU value[%d]: %d\n",m, images_out_gpu_serial[(i*IMG_WIDTH*IMG_HEIGHT) + m]);
				bad = 1;
				break;
			}
		}
		printf("\n%s\n", (bad) ? "Failed" : "Success");

    }
	
	hipFree(image_in);
	hipFree(image_out);
	//////////////////////////TODO: remove later
	hipFree(temp_histogram);
	hipFree(temp_cdf);
	
    t_finish = get_time_msec(); //Do not change
    distance_sqr = distance_sqr_between_image_arrays(images_out_cpu, images_out_gpu_serial); // Do not change
    printf("total time %f [msec]  distance from baseline %lld (should be zero)\n", t_finish - t_start, distance_sqr); //Do not change

    // GPU bulk
    printf("\n=== GPU Bulk ===\n"); //Do not change
    //TODO: allocate GPU memory for a all input images and all output images
    t_start = get_time_msec(); //Do not change
    //TODO: copy all input images from images_in to the GPU memory you allocated
    //TODO: invoke a kernel with N_IMAGES threadblocks, each working on a different image
    //TODO: copy output images from GPU memory to images_out_gpu_bulk
	CUDA_CHECK( hipMalloc((void **)&image_in, N_IMAGES * IMG_HEIGHT * IMG_WIDTH) );
    CUDA_CHECK( hipMalloc((void **)&image_out, N_IMAGES * IMG_HEIGHT * IMG_WIDTH) );
	hipMemcpy(image_in, images_in, N_IMAGES*IMG_WIDTH*IMG_HEIGHT, hipMemcpyDefault);
	process_image_kernel <<< N_IMAGES, NUM_THREADS >>> (image_in, image_out, temp_histogram, temp_cdf);   
	hipMemcpy(images_out_gpu_bulk, image_out, N_IMAGES*IMG_WIDTH*IMG_HEIGHT, hipMemcpyDefault);	
	hipFree(image_in);
	hipFree(image_out);
	
    t_finish = get_time_msec(); //Do not change
    distance_sqr = distance_sqr_between_image_arrays(images_out_cpu, images_out_gpu_bulk); // Do not change
    printf("total time %f [msec]  distance from baseline %lld (should be zero)\n", t_finish - t_start, distance_sqr); //Do not chhange

    return 0;
}
